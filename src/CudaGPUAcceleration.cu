
#include <hip/hip_runtime.h>
#if defined(_WIN32) || defined(__WIN32__) || defined(WIN32) || defined(_WIN64) || defined(__WIN64__) || defined(WIN64)

#include "CudaGPUAcceleration.h"
#include <cuda_runtime.h>
#include <device_launch_parameters.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <chrono>

#define CUDA_LOG(fmt, ...) printf("NDI Plugin CUDA: " fmt "\n", ##__VA_ARGS__)

// CUDA error checking macro
#define CUDA_CHECK(call) \
    do { \
        cudaError_t error = call; \
        if (error != cudaSuccess) { \
            CUDA_LOG("CUDA error at %s:%d - %s", __FILE__, __LINE__, cudaGetErrorString(error)); \
            return false; \
        } \
    } while(0)

#define CUDA_CHECK_VOID(call) \
    do { \
        cudaError_t error = call; \
        if (error != cudaSuccess) { \
            CUDA_LOG("CUDA error at %s:%d - %s", __FILE__, __LINE__, cudaGetErrorString(error)); \
        } \
    } while(0)

// CUDA GPU Context structure
struct CudaGPUContext {
    int deviceId;
    cudaDeviceProp deviceProps;
    cudaStream_t stream;
    char deviceName[256];
    
    // Memory pools for better performance
    float* d_rgbaInput;
    unsigned char* d_uyvyOutput;
    unsigned short* d_hdrOutput;
    size_t allocatedInputSize;
    size_t allocatedUyvySize;
    size_t allocatedHdrSize;
};

// CUDA kernel for RGBA to UYVY conversion
__global__ void rgba_to_uyvy_kernel(
    const float4* __restrict__ rgbaInput,
    uchar4* __restrict__ uyvyOutput,
    int width,
    int height
) {
    int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2; // Process two pixels at a time
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    // Flip vertically: OpenFX uses bottom-left origin, NDI expects top-left
    int srcRow = height - 1 - y;
    
    // Get two adjacent pixels
    int srcIdx1 = srcRow * width + x;
    int srcIdx2 = srcRow * width + x + 1;
    
    float4 pixel1 = rgbaInput[srcIdx1];
    float4 pixel2 = (x + 1 < width) ? rgbaInput[srcIdx2] : pixel1;
    
    // Clamp values to [0, 1]
    pixel1.x = fmaxf(0.0f, fminf(1.0f, pixel1.x));
    pixel1.y = fmaxf(0.0f, fminf(1.0f, pixel1.y));
    pixel1.z = fmaxf(0.0f, fminf(1.0f, pixel1.z));
    
    pixel2.x = fmaxf(0.0f, fminf(1.0f, pixel2.x));
    pixel2.y = fmaxf(0.0f, fminf(1.0f, pixel2.y));
    pixel2.z = fmaxf(0.0f, fminf(1.0f, pixel2.z));
    
    // Convert to YUV using Rec.709 coefficients
    float y1 = 0.2126f * pixel1.x + 0.7152f * pixel1.y + 0.0722f * pixel1.z;
    float y2 = 0.2126f * pixel2.x + 0.7152f * pixel2.y + 0.0722f * pixel2.z;
    
    float avgR = (pixel1.x + pixel2.x) * 0.5f;
    float avgG = (pixel1.y + pixel2.y) * 0.5f;
    float avgB = (pixel1.z + pixel2.z) * 0.5f;
    
    float u = -0.1146f * avgR - 0.3854f * avgG + 0.5f * avgB;
    float v = 0.5f * avgR - 0.4542f * avgG - 0.0458f * avgB;
    
    // Scale to 8-bit and pack as UYVY
    int dstIdx = y * (width / 2) + (x / 2);
    uyvyOutput[dstIdx] = make_uchar4(
        (unsigned char)((u + 0.5f) * 255.0f),  // U
        (unsigned char)(y1 * 255.0f),          // Y1
        (unsigned char)((v + 0.5f) * 255.0f),  // V
        (unsigned char)(y2 * 255.0f)           // Y2
    );
}

// CUDA kernel for RGBA to HDR P216 conversion
__global__ void rgba_to_hdr_p216_kernel(
    const float4* __restrict__ rgbaInput,
    unsigned short* __restrict__ yPlaneOutput,
    unsigned short* __restrict__ uvPlaneOutput,
    int width,
    int height,
    float scale
) {
    int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2; // Process two pixels for 4:2:2
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    // Flip vertically: OpenFX uses bottom-left origin, NDI expects top-left
    int srcRow = height - 1 - y;
    
    // Read two RGBA pixels
    int srcIdx1 = srcRow * width + x;
    int srcIdx2 = srcRow * width + x + 1;
    
    float4 rgba1 = rgbaInput[srcIdx1];
    float4 rgba2 = (x + 1 < width) ? rgbaInput[srcIdx2] : rgba1;
    
    // Clamp to 0-1 range
    rgba1.x = fmaxf(0.0f, fminf(1.0f, rgba1.x));
    rgba1.y = fmaxf(0.0f, fminf(1.0f, rgba1.y));
    rgba1.z = fmaxf(0.0f, fminf(1.0f, rgba1.z));
    
    rgba2.x = fmaxf(0.0f, fminf(1.0f, rgba2.x));
    rgba2.y = fmaxf(0.0f, fminf(1.0f, rgba2.y));
    rgba2.z = fmaxf(0.0f, fminf(1.0f, rgba2.z));
    
    // Convert to YUV using Rec.2020 coefficients for HDR
    float y1 = 0.2627f * rgba1.x + 0.6780f * rgba1.y + 0.0593f * rgba1.z;
    float y2 = 0.2627f * rgba2.x + 0.6780f * rgba2.y + 0.0593f * rgba2.z;
    
    // Average chroma for 4:2:2 subsampling
    float avgR = (rgba1.x + rgba2.x) * 0.5f;
    float avgG = (rgba1.y + rgba2.y) * 0.5f;
    float avgB = (rgba1.z + rgba2.z) * 0.5f;
    
    float u = -0.1396f * avgR - 0.3604f * avgG + 0.5f * avgB;
    float v = 0.5f * avgR - 0.4598f * avgG - 0.0402f * avgB;
    
    // Convert to 16-bit limited range (ITU BT.2100)
    // Y: 16-bit limited range [4096, 60160] for 10-bit equivalent [64, 940]
    // UV: 16-bit limited range [4096, 61440] for 10-bit equivalent [64, 960]
    unsigned short y1_16 = (unsigned short)(4096 + y1 * 56064); // (60160-4096)
    unsigned short y2_16 = (unsigned short)(4096 + y2 * 56064);
    unsigned short u_16 = (unsigned short)(32768 + u * 28672); // Center + range
    unsigned short v_16 = (unsigned short)(32768 + v * 28672);
    
    // Store in P216 format (planar)
    int yIdx1 = y * width + x;
    int yIdx2 = y * width + x + 1;
    int uvIdx = (y * width + x) / 2; // 4:2:2 subsampling
    
    yPlaneOutput[yIdx1] = y1_16;
    if (x + 1 < width) {
        yPlaneOutput[yIdx2] = y2_16;
    }
    
    // Store U and V interleaved for 4:2:2
    uvPlaneOutput[uvIdx * 2] = u_16;     // U
    uvPlaneOutput[uvIdx * 2 + 1] = v_16; // V
}

bool cuda_gpu_is_available(void) {
    int deviceCount = 0;
    cudaError_t error = cudaGetDeviceCount(&deviceCount);
    
    if (error != cudaSuccess || deviceCount == 0) {
        CUDA_LOG("CUDA not available: %s", cudaGetErrorString(error));
        return false;
    }
    
    // Check if we have at least one device with compute capability 3.0+
    for (int i = 0; i < deviceCount; i++) {
        cudaDeviceProp props;
        if (cudaGetDeviceProperties(&props, i) == cudaSuccess) {
            if (props.major >= 3) {
                CUDA_LOG("Found CUDA device %d: %s (Compute %d.%d)", 
                        i, props.name, props.major, props.minor);
                return true;
            }
        }
    }
    
    CUDA_LOG("No suitable CUDA devices found (need compute capability 3.0+)");
    return false;
}

CudaGPUContextRef cuda_gpu_init(void) {
    CUDA_LOG("Initializing CUDA GPU acceleration...");
    
    if (!cuda_gpu_is_available()) {
        return nullptr;
    }
    
    CudaGPUContext* context = (CudaGPUContext*)malloc(sizeof(CudaGPUContext));
    if (!context) {
        CUDA_LOG("Failed to allocate CUDA context");
        return nullptr;
    }
    
    memset(context, 0, sizeof(CudaGPUContext));
    
    // Find the best GPU (highest compute capability)
    int deviceCount = 0;
    cudaGetDeviceCount(&deviceCount);
    
    int bestDevice = 0;
    int bestMajor = 0, bestMinor = 0;
    
    for (int i = 0; i < deviceCount; i++) {
        cudaDeviceProp props;
        if (cudaGetDeviceProperties(&props, i) == cudaSuccess) {
            if (props.major > bestMajor || 
                (props.major == bestMajor && props.minor > bestMinor)) {
                bestDevice = i;
                bestMajor = props.major;
                bestMinor = props.minor;
            }
        }
    }
    
    context->deviceId = bestDevice;
    
    // Set device
    if (cudaSetDevice(context->deviceId) != cudaSuccess) {
        CUDA_LOG("Failed to set CUDA device %d", context->deviceId);
        free(context);
        return nullptr;
    }
    
    // Get device properties
    if (cudaGetDeviceProperties(&context->deviceProps, context->deviceId) != cudaSuccess) {
        CUDA_LOG("Failed to get device properties");
        free(context);
        return nullptr;
    }
    
    strncpy_s(context->deviceName, sizeof(context->deviceName), 
              context->deviceProps.name, sizeof(context->deviceName) - 1);
    
    // Create CUDA stream for asynchronous operations
    if (cudaStreamCreate(&context->stream) != cudaSuccess) {
        CUDA_LOG("Failed to create CUDA stream");
        free(context);
        return nullptr;
    }
    
    CUDA_LOG("CUDA GPU acceleration initialized successfully");
    CUDA_LOG("Device: %s (Compute %d.%d)", 
            context->deviceName, 
            context->deviceProps.major, 
            context->deviceProps.minor);
    CUDA_LOG("Global Memory: %.1f MB", 
            context->deviceProps.totalGlobalMem / (1024.0f * 1024.0f));
    
    return context;
}

void cuda_gpu_shutdown(CudaGPUContextRef context) {
    if (!context) return;
    
    CUDA_LOG("Shutting down CUDA GPU acceleration...");
    
    // Free device memory
    if (context->d_rgbaInput) {
        CUDA_CHECK_VOID(cudaFree(context->d_rgbaInput));
    }
    if (context->d_uyvyOutput) {
        CUDA_CHECK_VOID(cudaFree(context->d_uyvyOutput));
    }
    if (context->d_hdrOutput) {
        CUDA_CHECK_VOID(cudaFree(context->d_hdrOutput));
    }
    
    // Destroy stream
    if (context->stream) {
        CUDA_CHECK_VOID(cudaStreamDestroy(context->stream));
    }
    
    // Reset device
    CUDA_CHECK_VOID(cudaDeviceReset());
    
    free(context);
}

bool cuda_gpu_convert_rgba_to_uyvy(CudaGPUContextRef context, 
                                   const float* rgbaData, 
                                   unsigned char* uyvyData,
                                   int width, 
                                   int height) {
    if (!context || !rgbaData || !uyvyData) {
        CUDA_LOG("Invalid parameters for RGBA to UYVY conversion");
        return false;
    }
    
    auto startTime = std::chrono::high_resolution_clock::now();
    
    CUDA_LOG("Starting CUDA RGBA->UYVY conversion (%dx%d)", width, height);
    
    size_t inputSize = width * height * 4 * sizeof(float);
    size_t outputSize = width * height * 2; // UYVY is 2 bytes per pixel
    
    // Allocate or reallocate device memory if needed
    if (context->allocatedInputSize < inputSize) {
        if (context->d_rgbaInput) {
            CUDA_CHECK(cudaFree(context->d_rgbaInput));
        }
        CUDA_CHECK(cudaMalloc(&context->d_rgbaInput, inputSize));
        context->allocatedInputSize = inputSize;
    }
    
    if (context->allocatedUyvySize < outputSize) {
        if (context->d_uyvyOutput) {
            CUDA_CHECK(cudaFree(context->d_uyvyOutput));
        }
        CUDA_CHECK(cudaMalloc(&context->d_uyvyOutput, outputSize));
        context->allocatedUyvySize = outputSize;
    }
    
    // Copy input data to device
    CUDA_CHECK(cudaMemcpyAsync(context->d_rgbaInput, rgbaData, inputSize, 
                              cudaMemcpyHostToDevice, context->stream));
    
    // Configure kernel launch parameters
    dim3 blockSize(16, 16);
    dim3 gridSize((width / 2 + blockSize.x - 1) / blockSize.x, 
                  (height + blockSize.y - 1) / blockSize.y);
    
    // Launch kernel
    rgba_to_uyvy_kernel<<<gridSize, blockSize, 0, context->stream>>>(
        (float4*)context->d_rgbaInput,
        (uchar4*)context->d_uyvyOutput,
        width,
        height
    );
    
    // Check for kernel launch errors
    cudaError_t kernelError = cudaGetLastError();
    if (kernelError != cudaSuccess) {
        CUDA_LOG("CUDA kernel launch failed: %s", cudaGetErrorString(kernelError));
        return false;
    }
    
    // Copy result back to host
    CUDA_CHECK(cudaMemcpyAsync(uyvyData, context->d_uyvyOutput, outputSize, 
                              cudaMemcpyDeviceToHost, context->stream));
    
    // Wait for completion
    CUDA_CHECK(cudaStreamSynchronize(context->stream));
    
    auto endTime = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
    
    CUDA_LOG("🚀 CUDA RGBA->UYVY conversion completed in %lld μs (%.2f ms)", 
           duration.count(), duration.count() / 1000.0);
    
    return true;
}

bool cuda_gpu_convert_rgba_to_hdr(CudaGPUContextRef context,
                                  const float* rgbaData,
                                  unsigned short* hdrData,
                                  int width,
                                  int height,
                                  float scale) {
    if (!context || !rgbaData || !hdrData) {
        CUDA_LOG("Invalid parameters for RGBA to HDR conversion");
        return false;
    }
    
    auto startTime = std::chrono::high_resolution_clock::now();
    
    CUDA_LOG("Starting CUDA RGBA->HDR conversion (%dx%d)", width, height);
    
    size_t inputSize = width * height * 4 * sizeof(float);
    size_t outputSize = width * height * 2 * sizeof(unsigned short); // P216 format
    
    // Allocate or reallocate device memory if needed
    if (context->allocatedInputSize < inputSize) {
        if (context->d_rgbaInput) {
            CUDA_CHECK(cudaFree(context->d_rgbaInput));
        }
        CUDA_CHECK(cudaMalloc(&context->d_rgbaInput, inputSize));
        context->allocatedInputSize = inputSize;
    }
    
    if (context->allocatedHdrSize < outputSize) {
        if (context->d_hdrOutput) {
            CUDA_CHECK(cudaFree(context->d_hdrOutput));
        }
        CUDA_CHECK(cudaMalloc(&context->d_hdrOutput, outputSize));
        context->allocatedHdrSize = outputSize;
    }
    
    // Copy input data to device
    CUDA_CHECK(cudaMemcpyAsync(context->d_rgbaInput, rgbaData, inputSize, 
                              cudaMemcpyHostToDevice, context->stream));
    
    // Configure kernel launch parameters
    dim3 blockSize(16, 16);
    dim3 gridSize((width / 2 + blockSize.x - 1) / blockSize.x, 
                  (height + blockSize.y - 1) / blockSize.y);
    
    // Calculate plane pointers for P216 format
    unsigned short* yPlane = context->d_hdrOutput;
    unsigned short* uvPlane = context->d_hdrOutput + (width * height);
    
    // Launch kernel
    rgba_to_hdr_p216_kernel<<<gridSize, blockSize, 0, context->stream>>>(
        (float4*)context->d_rgbaInput,
        yPlane,
        uvPlane,
        width,
        height,
        scale
    );
    
    // Check for kernel launch errors
    cudaError_t kernelError = cudaGetLastError();
    if (kernelError != cudaSuccess) {
        CUDA_LOG("CUDA HDR kernel launch failed: %s", cudaGetErrorString(kernelError));
        return false;
    }
    
    // Copy result back to host
    CUDA_CHECK(cudaMemcpyAsync(hdrData, context->d_hdrOutput, outputSize, 
                              cudaMemcpyDeviceToHost, context->stream));
    
    // Wait for completion
    CUDA_CHECK(cudaStreamSynchronize(context->stream));
    
    auto endTime = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
    
    CUDA_LOG("🚀 CUDA RGBA->HDR conversion completed in %lld μs (%.2f ms)", 
           duration.count(), duration.count() / 1000.0);
    
    return true;
}

const char* cuda_gpu_get_device_name(CudaGPUContextRef context) {
    if (!context) return "Unknown";
    return context->deviceName;
}

bool cuda_gpu_get_memory_info(CudaGPUContextRef context, size_t* free_mem, size_t* total_mem) {
    if (!context || !free_mem || !total_mem) return false;
    
    if (cudaSetDevice(context->deviceId) != cudaSuccess) {
        return false;
    }
    
    return cudaMemGetInfo(free_mem, total_mem) == cudaSuccess;
}

#endif // Windows 